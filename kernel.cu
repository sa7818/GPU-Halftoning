#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdio.h>

using namespace cv;
using namespace std;

int help(void)
{
	cout << "Usage: " << endl
		<< "./exp_1 [mode] [image directory]" << endl;
	return 0;
}

int main(int argc, char *argv[])
{

	Mat source_image = imread(argv[argc - 1], 0);
	Mat target_image;
	// function declaration
	Mat halftoning(Mat source_image);
	Mat reduceIntensity(Mat source_image, int grayLevel);
	Mat arith(Mat img1, Mat img2);

	if (strcmp(argv[1], "--halftone") == 0) {
		target_image = halftoning(source_image);
	}
	else if (strcmp(argv[1], "--reduce") == 0) {
		int grayLevel;
		cout << "Please specify the gray level you want" << endl;
		cin >> grayLevel;
		target_image = reduceIntensity(source_image, grayLevel);
	}
	else if (strcmp(argv[1], "--arithmetic") == 0) {
		Mat img1 = imread(argv[2], 0);
		target_image = arith(img1, source_image);
	}
	else return help();

	imshow("target image", target_image);
	waitKey();
	return 0;
}

Mat halftoning(Mat source_image)
{
	Mat target_image(source_image.rows * 3, source_image.cols * 3, CV_8UC1);
	// arrays used to generate halftone pattern
	uchar halftone_source[10][3][3] =
	{   //0
		{
			{ 0, 0, 0 },
			{ 0, 0, 0 },
			{ 0, 0, 0 }
		},
			//1
		{
			{ 0, 255, 0 },
			{ 0, 0, 0 },
			{ 0, 0, 0 }
		},
			//2
		{
			{ 0, 255, 0 },
			{ 0, 0, 0 },
			{ 0, 0, 255 }
		},
			//3
		{
			{ 255, 255, 0 },
			{ 0, 0, 0 },
			{ 0, 0, 255 }
		},
			//4
		{
			{ 255, 255, 0 },
			{ 0, 0, 0 },
			{ 255, 0, 255 }
		},
			//5
		{
			{ 255, 255, 255 },
			{ 0, 0, 0 },
			{ 255, 0, 255 }
		},
			//6
		{
			{ 255, 255, 255 },
			{ 0, 0, 255 },
			{ 255, 0, 255 }
		},
			//7
		{
			{ 255, 255, 255 },
			{ 0, 0, 255 },
			{ 255, 255, 255 }
		},
			//8
		{
			{ 255, 255, 255 },
			{ 255, 0, 255 },
			{ 255, 255, 255 }
		},
			//9
		{
			{ 255, 255, 255 },
			{ 255, 255, 255 },
			{ 255, 255, 255 }
		},

	};


	// generating halftone dots pattern
	Mat halftone_matrix[10];
	for (int i = 0; i <= 9; i++) {
		halftone_matrix[i] = Mat(3, 3, CV_8UC1, halftone_source[i]);
	}

	// speed up the process by moving function call outside the loop
	uint source_rows = source_image.rows,
		source_cols = source_image.cols;
	// creating halftoning image
	for (uint i = 0; i < source_rows - 1; i++) {
		for (uint j = 0; j < source_cols - 1; j++) {

			int pixel_intensity = source_image.at<uchar>(i, j) / 28.3;
			halftone_matrix[pixel_intensity].copyTo(
				target_image(Rect_<uint>(j * 3, i * 3, 3, 3)));

		}
	}
	return target_image;
}

Mat reduceIntensity(Mat source_image, int grayLevel)
{
	uchar divider = ceil(255 / (double)grayLevel);

	// calculate and redistribute indensity for each pixel
	for (uint i = 0; i < source_image.rows; i++) {
		for (uint j = 0; j < source_image.cols; j++) {

			source_image.at<uchar>(i, j) = (
				source_image.at<uchar>(i, j) / divider) * (255 / (grayLevel - 1));

		}
	}
	return source_image;
}

Mat arith(Mat img1, Mat img2) {

	// convert to 16 bit to prevent overflow
	img1.convertTo(img1, CV_16UC1);
	img2.convertTo(img2, CV_16UC1);
	int user_choice;

	// resize the smaller image to fit the larger one
	if (img1.rows * img1.cols >= img2.rows * img2.cols) {
		resize(img2, img2, img1.size());
	}
	else {
		resize(img1, img1, img2.size());
	}

	cout << "What do you want: " << endl
		<< "1. add two images" << endl
		<< "2, multiply" << endl
		<< "3. divide" << endl
		<< "4. minus" << endl;

	cin >> user_choice;

	switch (user_choice) {
	case 1:
		add(img1, img2, img1);
		img1 *= 257;
		break;
	case 2:
		multiply(img1, img2, img1);
		break;
	case 3:
		divide(img1, img2, img1);
		img1 *= 256;
		break;
	case 4:
		subtract(img1, img2, img1);
		img1 *= 257;
		break;
	default:
		cout << "bad input";
		break;
	}
	return img1;
}